#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  int Row = blockIdx.x*blockDim.x + threadIdx.x;
  int Col = blockIdx.y*blockDim.y + threadIdx.y;
  __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];
  float result = 0;
  int numK = (numAColumns-1)/TILE_WIDTH+1; //numAColumns/TILE_WIDTH; //
  for(int k = 0; k != numK; ++k) {
    if(Row < numARows && k*TILE_WIDTH+threadIdx.y < numAColumns) {
      tileA[threadIdx.x][threadIdx.y] = A[Row*numAColumns + k*TILE_WIDTH+threadIdx.y];
    }
    if(k*TILE_WIDTH+threadIdx.x < numBRows && Col < numBColumns) {
      tileB[threadIdx.x][threadIdx.y] = B[(k*TILE_WIDTH+threadIdx.x)*numBColumns + Col];
    }
    __syncthreads();
    if(Row < numCRows && Col < numCColumns) {
      for(int m = 0; m != TILE_WIDTH; ++m) {
        if(k*TILE_WIDTH+m < numAColumns) {
          result += tileA[threadIdx.x][m]*tileB[m][threadIdx.y];
        }
      }
    }
    __syncthreads();
  }
  if(Row < numCRows && Col < numCColumns) {
    C[Row*numCColumns + Col] = result;
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  wbCheck(hipMalloc((void**) &deviceA, numARows*numAColumns*sizeof(float)));
  wbCheck(hipMalloc((void**) &deviceB, numBRows*numBColumns*sizeof(float)));
  wbCheck(hipMalloc((void**) &deviceC, numCRows*numCColumns*sizeof(float)));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, numARows*numAColumns*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, numBRows*numBColumns*sizeof(float), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid((numCRows-1)/TILE_WIDTH+1, (numCColumns-1)/TILE_WIDTH+1, 1);
  dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<DimGrid, DimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows*numCColumns*sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
